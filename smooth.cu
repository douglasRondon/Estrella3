#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

typedef struct
{
	unsigned char r;
	unsigned char g;
	unsigned char b;
}rgb;

typedef struct
{
	int columns;
	int size;
	int flag;
}info;

__global__ void smooth (rgb *, rgb *, int , int);
void hipError_t(hipError_t);


int main (int argc, char **argv){
	FILE *file;
	int i, rows, columns, max;
	rgb *imgH, *newImgH, *imgD, *newImgD;
    clock_t cInit, cFinal;
	
	file = fopen("2.ppm", "rb");
	fseek(file, 2, SEEK_SET);
	fscanf(file, "%d", &columns);
	fscanf(file, "%d", &rows);
	/* alocando memória para a matriz que irá armazenar as componentes r,g e b da imagem de entrada*/
	imgH = (rgb*) malloc ((rows*columns)*sizeof(rgb));
	
	/* alocando memória para a matriz que irá armazenar as componentes r,g e b da imagem de saída*/
	newImgH = (rgb*) malloc ((rows*columns)*sizeof(rgb));
	fscanf(file,"%d\n",&max);
	/* lendo a imagem do arquivo de entrada para a matriz */	
	for(i = 0; i < rows*columns; i++){
		fread(&imgH[i].r,sizeof(unsigned char),1,file);
		fread(&imgH[i].g,sizeof(unsigned char),1,file);
		fread(&imgH[i].b,sizeof(unsigned char),1,file);
	}
	fclose(file);
	
    cInit = clock(); /* COLOCA AQUI OU DEPOIS DO MALLOC???? */
	
	hipError_t(hipMalloc(&imgD, sizeof(rgb)*rows*columns));
	hipError_t(hipMalloc(&newImgD, sizeof(rgb)*rows*columns));
	hipError_t(hipMemcpy(imgD, imgH, sizeof(rgb)*rows*columns ,hipMemcpyHostToDevice));
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks ((columns + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1 ) / threadsPerBlock.y);
	
	smooth<<<numBlocks, threadsPerBlock>>>(imgD, newImgD, columns, rows);
	hipError_t(hipDeviceSynchronize());
	hipError_t(hipMemcpy(newImgH, newImgD, sizeof(rgb)*rows*columns ,hipMemcpyDeviceToHost));
    
    cFinal = clock();
	
    printf("Tempo: %lf segundos\n", (double)(cFinal - cInit) / CLOCKS_PER_SEC);

	/*criando a nova imagem */
	file = fopen("out.ppm", "wb");
	fprintf(file, "P6\n");
	fprintf(file, "%d %d\n",columns,rows);
	fprintf(file, "%d\n",max);
	for(i = 0; i < rows*columns; i++){
		fwrite(&newImgH[i].r ,sizeof(unsigned char),1,file);
		fwrite(&newImgH[i].g ,sizeof(unsigned char),1,file);
		fwrite(&newImgH[i].b ,sizeof(unsigned char),1,file);
	}
	fclose(file);

	/* liberando a memória utilizada */ 
	free(imgH);
	free(newImgH);
	hipFree(imgD);
	hipFree(newImgD);	
	return 0;	
}

/* função que retorna a média de uma componente do pixel utilizando os valores da componente ao redor dela (numa sub matriz 5x5) */
__global__ void smooth(rgb *image, rgb *newImg, int cols, int rows){
	int x, y;
	x = blockIdx.y * blockDim.y + threadIdx.y;
	y = blockIdx.x * blockDim.x + threadIdx.x;
	if(x > rows-1 || y > cols - 1)
		return;
	int i, j;
	int sumR = 0,sumG = 0,sumB = 0, count = 0;
	for(i = x-2; i < x+2; i++){
		for(j = y-2; j < y+2; j++){
			if((j < 0 || j > cols-1) || (i < 0 || i > rows-1));
			else{				
				sumR += image[i * cols + j].r;
				sumG += image[i * cols + j].g;
				sumB += image[i * cols + j].b;
				count++;
			}
		}
	}
	newImg[x * cols + y].r = sumR/count;
	newImg[x * cols + y].g = sumG/count;
	newImg[x * cols + y].b = sumB/count;
}

void hipError_t(hipError_t error){
	if (error != hipSuccess) {
		fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}